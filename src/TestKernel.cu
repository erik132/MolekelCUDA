#include "hip/hip_runtime.h"
#include "TestKernel.h"
#include "ESLogger.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>


__global__ void addKernel(int *result, int adder){
	
	atomicAdd(result,adder);
}

void activateCuda(){
	ESLogger esl("cudaLog.txt");
	int result=0,*d_result;
	char buffer[50];
	esl.logMessage("cuda activated");

	hipMalloc(&d_result,sizeof(int));
	hipMemcpy(d_result,&result,sizeof(int),hipMemcpyHostToDevice);
	addKernel<<<1,128>>>(d_result,2);
	hipMemcpy(&result,d_result,sizeof(int),hipMemcpyDeviceToHost);
	hipFree(d_result);
	
	sprintf(buffer, "cuda result was %d", result);
	esl.logMessage(buffer);
	

}

