#include "hip/hip_runtime.h"
#include "CalcDensCalcPoint.cuh"

#include "molekelHelpFunctions/CalcChiCalcPoint.cu"
#include "gputimer.h"

__global__ void calcPoint(CudaMolecule *molecule, CalcDensInternalData internalData, CudaMolecularOrbital *orbital, double *results){

	double result = 0;
	int indexZ = threadIdx.z + (blockDim.z*blockIdx.z);
	int	indexY = threadIdx.y + (blockDim.y*blockIdx.y);
	int	indexX = threadIdx.x + (blockDim.x*blockIdx.x);
	float x,y,z;

	if(indexX < internalData.ncub0 && indexY < internalData.ncub1 && indexZ < internalData.ncub2){
		
		x = internalData.dim0 + indexX*internalData.dx;
		y = internalData.dim2 + indexY*internalData.dy;
		z = internalData.dim4 + indexZ*internalData.dz;
		
		result = calcChiCalcPoint(orbital, molecule, x, y, z);
		results[indexX + (internalData.ncub0*indexY) + (internalData.ncub0*internalData.ncub1*indexZ)] = result;
	}
	
}

hipError_t CalcDensCalcPoint::initData(){
	ESLogger esl("CudaCalcDensCalcPoint.txt");
	char buffer[100];
	hipError_t status;

	resultsLength = calcData.ncub0*calcData.ncub1*calcData.ncub2;
	results = new double[resultsLength];
	
	status = CalcDensCalcPoint::moleculeToDevice();
	if(status != hipSuccess){
		esl.logMessage("Molecule copy failed");
		return status;
	}

	status = CalcDensCalcPoint::orbitalToDevice();
	if(status != hipSuccess){
		esl.logMessage("Orbital copy failed");
		return status;
	}

	status=hipMalloc((void**)&deviceResults, sizeof(double)*resultsLength);
	if(status != hipSuccess){
		sprintf(buffer, "memory allocation on device failed, errorcode %s", hipGetErrorString(status));
		esl.logMessage(buffer);
		return status;
	}

	return hipSuccess;
}

vtkImageData* CalcDensCalcPoint::runComputation(){
	
	ESLogger esl("CudaCalcDensCalcPoint.txt");
	hipError_t status;
	int i, j, k, counter;
	char buffer[100];
	vtkImageData* imageData;
	
	dim3 blockSize(BLOCK_DIM,BLOCK_DIM,BLOCK_DIM);
	dim3 gridSize = getGridSize();

	
	calcPoint<<<gridSize, blockSize>>>(deviceMolecule, calcData, deviceOrbital, deviceResults);

	status = hipGetLastError();
	if(status != hipSuccess){
		sprintf(buffer, "Kernel launch failed, errorcode %s", hipGetErrorString(status));
		esl.logMessage(buffer);
	}
	status = hipDeviceSynchronize();
	if(status != hipSuccess){
		sprintf(buffer, "Device synchronization failed, errorcode %s", hipGetErrorString(status));
		esl.logMessage(buffer);
	}

	status = hipMemcpy(results, deviceResults, sizeof(double)*resultsLength, hipMemcpyDeviceToHost);
	if(status != hipSuccess){
		sprintf(buffer, "memcpy from device failed, errorcode %s", hipGetErrorString(status));
		esl.logMessage(buffer);
	}
	
	imageData = initImageData();
	counter = 0;
	for (i=0; i<calcData.ncub2; i++) {
		for (j=0; j<calcData.ncub1; j++) {
			for (k=0; k<calcData.ncub0; k++) {
				imageData->SetScalarComponentFromDouble( k, j, i, 0, results[counter] );
				counter++;

			}
		}
	}

	return imageData;
}

void CalcDensCalcPoint::cleanupData(){
	CalcDensCalcPoint::deleteDeviceMoleculeData();
	CalcDensCalcPoint::deleteDeviceOrbitalData();
	hipFree(deviceResults);
	delete[] results;
}

CalcDensCalcPoint::CalcDensCalcPoint(CalcDensDataPack *data): CalcDensCudaFunction(data){
	
}


