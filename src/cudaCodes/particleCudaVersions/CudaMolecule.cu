#include "CudaMolecule.cuh"


void CudaMolecule::setProperties(Molecule *molecule){
	int i=0;

	nBasisFunctions = molecule->nBasisFunctions;
	atomsSize = molecule->Atoms.size();
	atoms = new CudaMolekelAtom[atomsSize];

	for(i=0; i<CudaMolecule::atomsSize; i++){
		atoms[i].setProperties(&molecule->Atoms[i]);
	}

}


CudaMolecule::~CudaMolecule(void){
	delete[] atoms;
}

hipError_t CudaMolecule::cpyInternalPointers(void){

	hipError_t status;
	int i;

	for(i=0; i<atomsSize; i++){
		status = atoms[i].cpyInternalPointers();
		if(status != hipSuccess){
			return status;
		}
	}
	
	status = hipMalloc((void**)&deviceAtoms,sizeof(CudaMolekelAtom)*atomsSize);
	if(status != hipSuccess){
		return status;
	}
	status = hipMemcpy(deviceAtoms, atoms, sizeof(CudaMolekelAtom)*atomsSize, hipMemcpyHostToDevice);

	return status;
}

void CudaMolecule::clearCudaData(void){
	int i;

	for(i=0; i<atomsSize; i++){
		atoms[i].clearCudaData();
	}

	hipFree(deviceAtoms);
}