#include "CudaShell.cuh"



void CudaShell::setProperties(Shell *shell){
	int i=0;
	
	gaussiansSize = shell->gaussians.size();
	nBase = shell->n_base;
	gaussians = new Gauss[gaussiansSize];

	for(i=0;i<gaussiansSize;i++){
		gaussians[i]=shell->gaussians[i];
	}

}

CudaShell::~CudaShell(void){
	delete[] gaussians;
};

hipError_t CudaShell::cpyInternalPointers(void){

	hipError_t status;
	
	status = hipMalloc((void**)&deviceGaussians, sizeof(Gauss)*gaussiansSize);
	if(status != hipSuccess){
		return status;
	}
	status = hipMemcpy(deviceGaussians, gaussians, sizeof(Gauss)*gaussiansSize, hipMemcpyHostToDevice);

	return status;
}

void CudaShell::clearCudaData(void){
	hipFree(deviceGaussians);
	//CudaShell::deviceGaussians = NULL;
}