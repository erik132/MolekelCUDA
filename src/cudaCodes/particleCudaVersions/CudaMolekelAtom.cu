#include "CudaMolekelAtom.cuh"



void CudaMolekelAtom::setProperties(MolekelAtom *atom){
	int i=0;

	for(int i=0;i<3;i++){
		coord[i] = atom->coord[i];
	}
	shellsSize = atom->Shells.size();
	shells = new CudaShell[shellsSize];

	for(i=0;i<shellsSize;i++){
		
		shells[i].setProperties(&atom->Shells[i]);
	}

}

CudaMolekelAtom::~CudaMolekelAtom(void){
	delete[] shells;

}

hipError_t CudaMolekelAtom::cpyInternalPointers(void){
	
	hipError_t status;
	int i;

	for(i=0; i<shellsSize; i++){
		status = shells[i].cpyInternalPointers();
		if(status != hipSuccess){
			return status;
		}
	}
	
	status = hipMalloc((void**)&deviceShells,sizeof(CudaShell)*shellsSize);
	if(status != hipSuccess){
		return status;
	}
	status = hipMemcpy(deviceShells, shells, sizeof(CudaShell)*shellsSize, hipMemcpyHostToDevice);

	return status;
}

void CudaMolekelAtom::clearCudaData(void){
	int i;

	for(i=0; i<shellsSize; i++){
		shells[i].clearCudaData();
	}

	hipFree(deviceShells);
	//CudaMolekelAtom::deviceShells = NULL;
}