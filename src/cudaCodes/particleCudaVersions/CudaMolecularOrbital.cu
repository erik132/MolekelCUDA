#include "CudaMolecularOrbital.cuh"


void CudaMolecularOrbital::setProperties(MolecularOrbital *molOrb, Molecule *mol){

	coefficients = molOrb->coefficient;
	coefficientsSize = mol->nBasisFunctions;
}


hipError_t CudaMolecularOrbital::cpyInternalPointers(void){
	hipError_t status;

	status = hipMalloc((void**)&deviceCoefficients,sizeof(double)*coefficientsSize);
	if(status!=hipSuccess){
		return status;
	}

	status = hipMemcpy(deviceCoefficients, coefficients, sizeof(double)*coefficientsSize, hipMemcpyHostToDevice);
	return status;
}

void CudaMolecularOrbital::clearCudaData(void){
	hipFree(deviceCoefficients);

}

CudaMolecularOrbital::~CudaMolecularOrbital(void){


}