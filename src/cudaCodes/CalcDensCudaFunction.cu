#include "CalcDensCudaFunction.cuh"


vtkImageData* CalcDensCudaFunction::initImageData(){

	vtkSmartPointer< vtkImageData > image( vtkImageData::New() );

	image->SetDimensions( calcData.ncub0, calcData.ncub1, calcData.ncub2 );
	image->SetOrigin( calcData.dim0,calcData.dim2,calcData.dim4 );
	image->SetSpacing( calcData.dx, calcData.dy, calcData.dz );

	return image;
}

CalcDensCudaFunction::CalcDensCudaFunction(CalcDensDataPack *data){
	
	ESLogger esl("CalcDensCudaFunction.txt");
	esl.logMessage("function started");
	BLOCK_DIM = 5;
	
	calcData.minValue = data->minValue;
	calcData.maxValue = data->maxValue;

	calcData.ncub0 = *data->ncubes++;
	calcData.ncub1 = *data->ncubes++;
	calcData.ncub2 = *data->ncubes++;

	calcData.dim0 = data->dim[0];
	calcData.dim2 = data->dim[2];
	calcData.dim4 = data->dim[4];

	calcData.dx = (data->dim[1] - data->dim[0]) / (calcData.ncub0 - 1);
	calcData.dy = (data->dim[3] - data->dim[2]) / (calcData.ncub1 - 1);
	calcData.dz = (data->dim[5] - data->dim[4]) / (calcData.ncub2 - 1);
	
	cudaMolecule.setProperties(data->mol);
	cudaOrbital.setProperties(data->orbital,data->mol);

	

	if(data->orbital == NULL){
		esl.logMessage("data pack molecularOrbital is NULL");
	}else{
		esl.logMessage("data pac molecularOrbital is not NULL");
	}
}



hipError_t CalcDensCudaFunction::moleculeToDevice(){
	
	hipError_t status;

	status = cpyOrbital();

	if(status != hipSuccess){
		return status;
	}


	status = cudaMolecule.cpyInternalPointers();

	if(status!=hipSuccess){
		return status;
	}

	status = hipMalloc((void**)&deviceMolecule, sizeof(CudaMolecule));

	if(status!=hipSuccess){
		return status;
	}
	status = hipMemcpy(deviceMolecule, &cudaMolecule, sizeof(CudaMolecule),hipMemcpyHostToDevice);

	return status;
}

void CalcDensCudaFunction::deleteDeviceData(){

	cudaMolecule.clearCudaData();
	hipFree(deviceMolecule);
	cudaOrbital.clearCudaData();
	hipFree(deviceOrbital);

}

hipError_t CalcDensCudaFunction::cpyOrbital(){

	hipError_t status = cudaOrbital.cpyInternalPointers();
	if(status != hipSuccess){
		return status;
	}

	status = hipMalloc((void**)&deviceOrbital, sizeof(CudaMolecularOrbital));

	if(status != hipSuccess){
		return status;
	}
	status = hipMemcpy(deviceOrbital, &cudaOrbital, sizeof(CudaMolecularOrbital),hipMemcpyHostToDevice);

	return status;
}

dim3 CalcDensCudaFunction::getGridSize(){
	
	int x,y,z; //size nr for each direction
	char buffer[1000];
	ESLogger esl("getGridSize.txt");
	
	x= getSingleGridSize(calcData.ncub0, BLOCK_DIM);
	y= getSingleGridSize(calcData.ncub1, BLOCK_DIM);
	z= getSingleGridSize(calcData.ncub2, BLOCK_DIM);

	sprintf(buffer, "grids will be: x: %d y: %d z: %d", x, y, z);
	esl.logMessage(buffer);

	dim3 gridSize(x,y,z);
	return gridSize;

}

int CalcDensCudaFunction::getSingleGridSize(int elements, int blockSize){
	int result;

	result = elements/blockSize;
	if(result*blockSize < elements){
		result++;
	}

	return result;
}