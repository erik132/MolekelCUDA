#include "hip/hip_runtime.h"
#include "CalcDensCalculateDensity.cuh"

#include "molekelHelpFunctions/CalcChiCalculateDensity.cu"

#include "gputimer.h"

__global__ void checkDensityMatrix(float* densities, double* resultArray, int densityLength){
	int i=0;
	//densities = densities + (idx*(idx+1))/2;

	for(i=0; i<=50; i++){
		resultArray[i] = densities[i];
	}
	
}

__global__ void calculateDensity(CudaMolecule *molecule, CalcDensInternalData internalData, CudaMolecularOrbital *orbital, float *densities, double *results){
	double result = 0;
	int indexZ = threadIdx.z + (blockDim.z*blockIdx.z);
	int	indexY = threadIdx.y + (blockDim.y*blockIdx.y);
	int	indexX = threadIdx.x + (blockDim.x*blockIdx.x);
	float x,y,z;

	if(indexX < internalData.ncub0 && indexY < internalData.ncub1 && indexZ < internalData.ncub2){
		
		x = internalData.dim0 + indexX*internalData.dx;
		y = internalData.dim2 + indexY*internalData.dy;
		z = internalData.dim4 + indexZ*internalData.dz;
		
		result = calcChiCalculateDensity(densities,orbital, molecule, x, y, z);
		results[indexX + (internalData.ncub0*indexY) + (internalData.ncub0*internalData.ncub1*indexZ)] = result;
	}
}



hipError_t CalcDensCalculateDensity::initData(){
	char buffer[100];
	hipError_t status;

	resultsLength = calcData.ncub0*calcData.ncub1*calcData.ncub2;
	results = new double[resultsLength];

	this->esl->logMessage("starting to init data");
	
	status = this->moleculeToDevice();
	if(status != hipSuccess){
		esl->logMessage("Molecule copy failed");
		return status;
	}

	status = this->orbitalToDevice();
	if(status != hipSuccess){
		esl->logMessage("Orbital copy failed");
		return status;
	}

	status=hipMalloc((void**)&deviceResults, sizeof(double)*resultsLength);
	if(status != hipSuccess){
		sprintf(buffer, "memory allocation on device failed, errorcode %s", hipGetErrorString(status));
		esl->logMessage(buffer);
		return status;
	}
	
	if(!this->createDensityMatrix()){
		this->esl->logMessage("can not create density matrix.");
		return hipErrorOutOfMemory;
	}
	

	status = this->densityMatrixToDevice();
	if(status != hipSuccess){
		sprintf(buffer, "density matrix allocation to device failed, errorcode %s", hipGetErrorString(status));
		this->esl->logMessage(buffer);
	}

	this->esl->logMessage("all operations on initdata finished as a success");
	return hipSuccess;
}

vtkImageData* CalcDensCalculateDensity::runComputation(){
	char buffer[100];
	hipError_t status;
	int i, j, k, counter=0;
	vtkImageData* imageData;


	dim3 blockSize(BLOCK_DIM,BLOCK_DIM,BLOCK_DIM);
	dim3 gridSize = getGridSize();

	sprintf(buffer, "Density matrix has %d elems and is %d bytes long",calcData.densityLength, sizeof(float)*calcData.densityLength);
	this->esl->logMessage(buffer);

	calculateDensity<<<gridSize, blockSize>>>(this->deviceMolecule,this->calcData,this->deviceOrbital,this->deviceDensityMatrix, this->deviceResults);

	status = hipGetLastError();
	if(status != hipSuccess){
		sprintf(buffer, "kernel launch failed, errorcode %s", hipGetErrorString(status));
		this->esl->logMessage(buffer);
		return NULL;
	}

	status = hipDeviceSynchronize();
	if(status != hipSuccess){
		sprintf(buffer, "failed to sync devices %s", hipGetErrorString(status));
		this->esl->logMessage(buffer);
		return NULL;
	}

	status = hipMemcpy(results,deviceResults,resultsLength*sizeof(double),hipMemcpyDeviceToHost);
	if(status != hipSuccess){
		sprintf(buffer, "results copy back to host failed, errorcode %s", hipGetErrorString(status));
		this->esl->logMessage(buffer);
		return NULL;
	}

	/*for(i=0; i<resultsLength; i++){
		sprintf(buffer, "result nr %d is %.15f", i, results[i]);
		this->esl->logMessage(buffer);
	}*/

	imageData = initImageData();
	counter = 0;
	for (i=0; i<calcData.ncub2; i++) {
		for (j=0; j<calcData.ncub1; j++) {
			for (k=0; k<calcData.ncub0; k++) {
				imageData->SetScalarComponentFromDouble( k, j, i, 0, results[counter] );
				counter++;

			}
		}
	}

	return imageData;
}

void CalcDensCalculateDensity::cleanupData(){
	this->esl->logMessage("starting to clean up.");

	this->deleteDeviceMoleculeData();
	this->deleteDeviceOrbitalData();
	this->deleteDeviceDensityMatrix();
	this->deleteDensityMatrix();
	hipFree(deviceResults);
	delete[] results;

	this->esl->logMessage("cleanup complete.");
}

CalcDensCalculateDensity::CalcDensCalculateDensity(CalcDensDataPack *data): CalcDensCudaFunction(data){
	this->esl = new ESLogger("CalcDensCalculateDensity.txt");
}

CalcDensCalculateDensity::~CalcDensCalculateDensity(){
	delete this->esl;
}