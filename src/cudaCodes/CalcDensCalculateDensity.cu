#include "hip/hip_runtime.h"
#include "CalcDensCalculateDensity.cuh"

#include "molekelHelpFunctions/CalcChi.cu"

#include "gputimer.h"

__global__ void checkDensityMatrix(float* densities, double* resultArray, int densityLength){
	int i=0;
	//densities = densities + (idx*(idx+1))/2;

	for(i=0; i<=50; i++){
		resultArray[i] = densities[i];
	}
	
}



hipError_t CalcDensCalculateDensity::initData(){
	char buffer[100];
	hipError_t status;

	resultsLength = calcData.ncub0*calcData.ncub1*calcData.ncub2;
	results = new double[resultsLength];

	this->esl->logMessage("starting to init data");
	
	status = this->moleculeToDevice();
	if(status != hipSuccess){
		esl->logMessage("Molecule copy failed");
		return status;
	}

	status = this->orbitalToDevice();
	if(status != hipSuccess){
		esl->logMessage("Orbital copy failed");
		return status;
	}

	status=hipMalloc((void**)&deviceResults, sizeof(double)*resultsLength);
	if(status != hipSuccess){
		sprintf(buffer, "memory allocation on device failed, errorcode %s", hipGetErrorString(status));
		esl->logMessage(buffer);
		return status;
	}
	
	if(!this->createDensityMatrix()){
		this->esl->logMessage("can not create density matrix.");
		return hipErrorOutOfMemory;
	}
	

	status = this->densityMatrixToDevice();
	if(status != hipSuccess){
		sprintf(buffer, "density matrix allocation to device failed, errorcode %s", hipGetErrorString(status));
		this->esl->logMessage(buffer);
	}

	this->esl->logMessage("all operations on initdata finished as a success");
	return hipSuccess;
}

vtkImageData* CalcDensCalculateDensity::runComputation(){
	char buffer[100];
	dim3 blockSize(1,1,1);
	dim3 gridSize(1,1,1);
	hipError_t status;
	int i=0;

	sprintf(buffer, "Density matrix has %d elems and is %d bytes long",this->densityMatrixLength, sizeof(float)*this->densityMatrixLength);
	this->esl->logMessage(buffer);

	checkDensityMatrix<<<blockSize, gridSize>>>(this->deviceDensityMatrix,this->deviceResults,this->densityMatrixLength);

	status = hipGetLastError();
	if(status != hipSuccess){
		sprintf(buffer, "kernel launch failed, errorcode %s", hipGetErrorString(status));
		this->esl->logMessage(buffer);
		return NULL;
	}

	status = hipDeviceSynchronize();
	if(status != hipSuccess){
		sprintf(buffer, "failed to sync devices %s", hipGetErrorString(status));
		this->esl->logMessage(buffer);
		return NULL;
	}

	status = hipMemcpy(results,deviceResults,resultsLength*sizeof(double),hipMemcpyDeviceToHost);
	if(status != hipSuccess){
		sprintf(buffer, "results copy back to host failed, errorcode %s", hipGetErrorString(status));
		this->esl->logMessage(buffer);
		return NULL;
	}

	for(i=0; i<this->densityMatrixLength; i++){
		sprintf(buffer, "nr %d is %f", i, this->results[i]);
		this->esl->logMessage(buffer);
	}

	return NULL;
}

void CalcDensCalculateDensity::cleanupData(){
	this->esl->logMessage("starting to clean up.");

	this->deleteDeviceMoleculeData();
	this->deleteDeviceOrbitalData();
	this->deleteDeviceDensityMatrix();
	this->deleteDensityMatrix();
	hipFree(deviceResults);
	delete[] results;

	this->esl->logMessage("cleanup complete.");
}

CalcDensCalculateDensity::CalcDensCalculateDensity(CalcDensDataPack *data): CalcDensCudaFunction(data){
	this->esl = new ESLogger("CalcDensCalculateDensity.txt");
}

CalcDensCalculateDensity::~CalcDensCalculateDensity(){
	delete this->esl;
}