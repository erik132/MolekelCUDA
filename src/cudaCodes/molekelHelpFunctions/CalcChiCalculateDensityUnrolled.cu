#include "hip/hip_runtime.h"
#pragma once

#include "../particleCudaVersions/CudaMolecule.cuh"
#include "../particleCudaVersions/CudaMolecularOrbital.cuh"

#include "../../old/constant.h"
#include "../../old/molekeltypes.h"

#include "hip/hip_runtime.h"
#include ""

static __device__ double calcChiCalculateDensityUnrolled(float * densities, CudaMolecularOrbital *orbital, CudaMolecule *molecule,const float x,const float y,const float z,const int densityLength,int rowNr){

	
	double radial_part;
	float xa, ya, za, ra2;  /* atomic units !! */
	int atom, shell, gauss, i;
	int	rowElem=0; //which element are we currently refering to
	int densityIndex = 0;

	const int atomsSize = molecule->atomsSize;
	int shellsSize, gaussSize, count=0;
	Gauss tempGauss;

	double cp[10], result = 0, tempResult = 0; //temporary cells will be used to gather gaussian calculations in them to later add them to the result.
	
	densityIndex = (rowNr * (rowNr+1))/2;

	for(i=0; i<10; i++){
		cp[i] = 0;
	}
	
	for (atom=0; atom<atomsSize; atom++) {
		xa = (x - molecule->deviceAtoms[atom].coord[0]) * _1_BOHR;
		ya = (y - molecule->deviceAtoms[atom].coord[1]) * _1_BOHR;
		za = (z - molecule->deviceAtoms[atom].coord[2]) * _1_BOHR;

		ra2 = xa*xa + ya*ya + za*za;    /* cutoff-distance ? */
		shellsSize=molecule->deviceAtoms[atom].shellsSize;

		for (shell=0; shell<shellsSize; shell++) {
			gaussSize = molecule->deviceAtoms[atom].deviceShells[shell].gaussiansSize;
			switch(molecule->deviceAtoms[atom].deviceShells[shell].nBase){
			case 1  :        /*** S-orbital ***/
				for (gauss=0; gauss<gaussSize; gauss++) {
					tempGauss = molecule->deviceAtoms[atom].deviceShells[shell].deviceGaussians[gauss];
					radial_part = exp(-ra2*tempGauss.exponent);
					*cp += tempGauss.coeff * radial_part;
				}
				
				count = 1;
				break;

			case 4 :        /*** SP-orbital ***/
				for (gauss=0; gauss<gaussSize; gauss++) {
					tempGauss = molecule->deviceAtoms[atom].deviceShells[shell].deviceGaussians[gauss];
					radial_part = exp(-ra2*tempGauss.exponent);
					*cp     += tempGauss.coeff * radial_part;
					*(cp+1) += tempGauss.coeff2 * xa * radial_part;
					*(cp+2) += tempGauss.coeff2 * ya * radial_part;
					*(cp+3) += tempGauss.coeff2 * za * radial_part;
				}
				
				count = 4;
				break;

			case 3  :        /*** P-orbital ***/
				for (gauss=0; gauss<gaussSize; gauss++) {
					tempGauss = molecule->deviceAtoms[atom].deviceShells[shell].deviceGaussians[gauss];
					radial_part = tempGauss.coeff * exp(-ra2*tempGauss.exponent);
					*cp     += xa * radial_part;
					*(cp+1) += ya * radial_part;
					*(cp+2) += za * radial_part;
					}
				
				count = 3;
				break;

			case 5  :        /*** D-orbital (5) ***/
				for (gauss=0; gauss<gaussSize; gauss++) {
					tempGauss = molecule->deviceAtoms[atom].deviceShells[shell].deviceGaussians[gauss];
					radial_part = tempGauss.coeff * exp(-ra2*tempGauss.exponent);
					*cp    += 0.288675135 *
						   (2*za*za - xa*xa - ya*ya) * radial_part;
					*(cp+3) += 0.5 * (xa*xa - ya*ya) * radial_part;
					*(cp+4) += xa * ya * radial_part;
					*(cp+1) += xa * za * radial_part;
					*(cp+2) += ya * za * radial_part;
				}
				count = 5;
				break;

			case 6  :        /*** D-orbital (6) ***/
				for (gauss=0; gauss<gaussSize; gauss++) {
					tempGauss = molecule->deviceAtoms[atom].deviceShells[shell].deviceGaussians[gauss];
					radial_part = tempGauss.coeff * exp(-ra2*tempGauss.exponent);
					*cp     += radial_part * xa * xa * 0.57735027;
					*(cp+1) += radial_part * ya * ya * 0.57735027;
					*(cp+2) += radial_part * za * za * 0.57735027;
					*(cp+3) += radial_part * xa * ya;
					*(cp+4) += radial_part * xa * za;
					*(cp+5) += radial_part * ya * za;
				}
				count = 6;
				break;

			case 7  :        /*** F-orbital (7) ***/
				for (gauss=0; gauss<gaussSize; gauss++) {
					tempGauss = molecule->deviceAtoms[atom].deviceShells[shell].deviceGaussians[gauss];
					radial_part = exp(-ra2*tempGauss.exponent) * tempGauss.coeff;
					*cp     += radial_part * za * (5. * za * za - 3. * ra2)/* * k */;
					*(cp+1) += radial_part * xa * (5. * za * za - ra2)/* * k */;
					*(cp+2) += radial_part * ya * (5. * za * za - ra2)/* * k */;
					*(cp+3) += radial_part * za * (xa * xa - ya * ya)/* * k */;
					*(cp+4) += radial_part * xa * ya * za;
					*(cp+5) += radial_part * (xa * xa * xa - 3. * xa * ya * ya)/* * k */;
					*(cp+6) += radial_part * (3. * xa * xa * ya - ya * ya * ya)/* * k */;
				}
				count = 7;
				break;

			case 10 :        /*** F-orbital (10) ***/
				   /* correct order ??? */
				for (gauss=0; gauss<gaussSize; gauss++) {
					tempGauss = molecule->deviceAtoms[atom].deviceShells[shell].deviceGaussians[gauss];
					radial_part = tempGauss.coeff * exp(-ra2*tempGauss.exponent);
					*cp     += radial_part * xa * xa * xa * .25819889;
					*(cp+1) += radial_part * ya * ya * ya * .25819889;
					*(cp+2) += radial_part * za * za * za * .25819889;
					*(cp+3) += radial_part * xa * xa * ya * .57735027;
					*(cp+4) += radial_part * xa * xa * za * .57735027;
					*(cp+5) += radial_part * xa * ya * ya * .57735027;
					*(cp+6) += radial_part * ya * ya * za * .57735027;
					*(cp+7) += radial_part * xa * za * za * .57735027;
					*(cp+8) += radial_part * ya * za * za * .57735027;
					*(cp+9) += radial_part * xa * ya * za;
				}
				
				count = 10;
				break;

			} /* end of switch */
			

			for(i=0; i<count; i++){
				if(rowElem == rowNr){
					result += cp[i] * tempResult;
					result += densities[densityIndex] * cp[i] * cp[i];
					rowElem=0;
					densityIndex++;
					tempResult = 0;
					break;
				}else{
					tempResult += densities[densityIndex] * cp[i] * 2.0;
					rowElem++;
					densityIndex++;
				}
			}
			
			if(rowElem == 0){
				break;
			}
			for(i=0; i<count; i++){
				cp[i] = 0;
			}
			count = 0;
			
		} /* end of loop over the shells (for(sp...) */
		if(rowElem == 0){
			break;
		}
	} /* end of loop over the atoms (for(ap...)*/
	return result;
}